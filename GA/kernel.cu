#include "hip/hip_runtime.h"
// thrust library
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
using namespace std;

const unsigned int N = 3;
const unsigned int GENERATION_SIZE = 100;
const unsigned int start_xd_min[N] = { 2112,1180,417 };
unsigned int individuals[N][GENERATION_SIZE] = {};

#include "cpu_rnd.cuh"
#include "cuda_rnd.cuh"

#include "compute_time.cuh"

const int n = 3;
const int ns = 2;
const int allNodes = n + ns;
int LT[allNodes][n] = {};

const int x_inf = 100000;
const int simTime = 60;

void gpu()
{
	// device storage for doubles
	thrust::device_vector<double> population(N * GENERATION_SIZE);

	// Fill the vector with random distribution (0,1)
	thrust::counting_iterator<unsigned int> index_sequence_begin(0);
	thrust::transform(index_sequence_begin,
		index_sequence_begin + GENERATION_SIZE * N,
		population.begin(),
		cuda_rnd());


	thrust::device_vector<int> LTcuda(allNodes * n);

	thrust::copy(&(LT[0][0]), &(LT[allNodes - 1][n - 1]), LTcuda.begin());
	thrust::device_vector<int>::iterator iter =
		thrust::max_element(LTcuda.begin(), LTcuda.end());
	int Lcuda = *iter;

	ComputeTimeStart();
	for (int node = 0; node < N; node++) {
		for (int individualNo = 0; individualNo < GENERATION_SIZE; individualNo++) {
			int index = individualNo + node * GENERATION_SIZE;
			population[index] = round(population[index] * start_xd_min[node]);
		}
	}

	// print contents of population
	for (int i = 0; i < population.size(); i++) {
		if (i % GENERATION_SIZE == 0) {
			cout << "\n\nNODE\n";
		}
		cout << population[i] << " ";
	}

	double tt = ComputeTimeEnd();
	cout << "\nGPU time: " << tt << " ms\n";
}

#include "vector_types.cuh"
#include "vector_operators.cuh"
#include "vector_operations.cuh"

void cpu()
{
	ComputeTimeStart();
	for (int node = 0; node < N; node++) {
		for (int individualNo = 0; individualNo < GENERATION_SIZE; individualNo++) {
			individuals[node][individualNo] = static_cast<int>(round(rnd() * start_xd_min[node]));
		}
	}

	for (int node = 0; node < N; node++) {
		cout << "\nNODE\n";
		for (int individualNo = 0; individualNo < GENERATION_SIZE; individualNo++) {
			cout << individuals[node][individualNo] << " ";
		}
		cout << "\n";
	}

	LT[3][0] = 2;
	LT[4][1] = 4;
	LT[1][2] = 3;
	LT[0][2] = 3;
	LT[0][1] = 1;

	int* start = &LT[0][0];
	// max lead time
	int L = *max_element(start, start + allNodes * n);

	float LA_nom[allNodes][n] = {};
	three_dimension_vector_float LA(simTime, vector<vector<float>>(allNodes, vector<float>(n)));

	LA[0][3][0] = LA_nom[3][0] = 1;
	LA[0][4][1] = LA_nom[4][1] = 0.8f;
	LA[0][1][2] = LA_nom[1][2] = 0.6f;
	LA[0][0][2] = LA_nom[0][2] = 0.4f;
	LA[0][0][1] = LA_nom[0][1] = 0.2f;
	//Print2DVector<float>(LA[0], "delay matrix");

	// Verify if allocation correct - elements in each column should sum up to 1 or 0	
	for (int j = 0; j < n; j++) {
		float temp = 0;
		for (int i = 0; i < allNodes; i++) {
			temp = temp + LA[0][i][j];
		}

		if (temp != 0 && temp != 1) {
			throw ("Improper allocation in column: %d", j);
		}
	}

	// Initial conditions
	int time[simTime] = {};
	int u[n][simTime] = {};
	int u_hist[n][simTime] = {}; // order history
	two_dimension_vector_int x(simTime, vector<int>(allNodes, 0));
	int y[allNodes][simTime] = {};
	int xd[allNodes] = { 80, 120, 98, 0, 0 };

	x[0] = { 70, 140, 88, x_inf, x_inf }; // initial stock level
	//Print2DVector<int>(x, "stock level ");

	// Demand
	int dmax[n] = { 10, 15, 20 };
	two_dimension_vector_int d(n, vector<int>(simTime, {})); // int d[n][simTime] = {};

	for (int j = 0; j < simTime; j++) {
		double multiplier = 1;
		for (int k = 0; k < n; k++) {
			double demand = multiplier * dmax[k] * rnd();
			int randomDemand = static_cast<int>(round(demand));
			if (randomDemand > dmax[k]) d[k][j] = dmax[k];
			else d[k][j] = randomDemand;
		}
	};
	//Print2DVector<int>(d, "demand");

	// State - space description
	// System matrices
	three_dimension_vector_float B_nom(L, two_dimension_vector_float(n, vector<float>(n)));
	four_dimension_vector_float B(simTime, three_dimension_vector_float(L, two_dimension_vector_float(n, vector<float>(n))));

	// Assuming zero order processing time(eq 9)
	two_dimension_vector_float B_0(n, vector<float>(n));
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			B_0[i][j] = LA[0][i][j] * -1;
		}
	}

	// index k corresponds to delay k(eq 8)	
	for (int k = 0; k < L; k++) {
		for (int j = 0; j < n; j++) {
			float t_sum = 0;
			for (int i = 0; i < allNodes; i++) {
				if (LT[i][j] == k + 1) {
					t_sum = t_sum + LA[0][i][j];
				}
			}
			B_nom[k][j][j] = t_sum;
		}
	}
	//Print3DVector<float>(B_nom, "B matrix");
	B[0] = B_nom;

	//% Sum of delay matrices
	two_dimension_vector_float Lambda(n, vector<float>(n, {}));

	// table index k corresponds to delay k
	for (int k = 0; k < L; k++) {
		Lambda = Lambda + B[0][k];
	}

	Lambda = Lambda + B_0; // eq 11 

	double tt = ComputeTimeEnd();
	cout << "CPU time: " << tt << " ms\n";
}

int main()
{
	cpu();
	gpu();

	return 0;
}