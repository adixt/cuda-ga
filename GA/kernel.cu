#include "hip/hip_runtime.h"
#include <iostream>
#include <Windows.h>
#include "hip/hip_runtime.h"
#include ""
#include <ctime>
#include <cmath>
#include <random>
#include <functional>

// thrust library
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>

using namespace std;

const unsigned int N = 3;
const unsigned int GENERATION_SIZE = 100;
const int start_xd_min[N] = { 2112,1180,417 };
int individuals[N][GENERATION_SIZE] = {};

// uniform distribution
default_random_engine generator;
uniform_real_distribution<double> distribution(0, 1);
double dice_roll = distribution(generator);
auto rnd = bind(distribution, generator);

/* Generate CUDA uniform distribution
 *
 * Used by thrust transform functions to create large numbers of
 * random numbers in a uniform distribution.
 */
struct cuda_rnd
{
	double a, b;

	__host__ __device__
		cuda_rnd(double _a = 0.f, double _b = 1.f) : a(_a), b(_b) {
	};

	__host__ __device__
		double operator()(const unsigned int n) const
	{
		thrust::default_random_engine rng;
		thrust::uniform_real_distribution<double> dist(a, b);
		rng.discard(n);

		return dist(rng);
	}
};

LARGE_INTEGER tb, te, tf;

void ComputeTimeStart()
{
	QueryPerformanceFrequency(&tf);
	QueryPerformanceCounter(&tb);
}

double ComputeTimeEnd()
{
	QueryPerformanceCounter(&te);
	return 1000.0 * (double(te.QuadPart - tb.QuadPart)) / double(tf.QuadPart);
}

void gpu()
{
	// device storage for doubles
	thrust::device_vector<double> population(N * GENERATION_SIZE);

	// Fill the vector with random distribution (0,1)
	thrust::counting_iterator<unsigned int> index_sequence_begin(0);
	thrust::transform(index_sequence_begin,
		index_sequence_begin + GENERATION_SIZE * N,
		population.begin(),
		cuda_rnd());

	ComputeTimeStart();
	for (int node = 0; node < N; node++) {
		for (int individualNo = 0; individualNo < GENERATION_SIZE; individualNo++) {
			int index = individualNo + node * GENERATION_SIZE;
			population[index] = round(population[index] * start_xd_min[node]);
		}
	}

	// print contents of population
	for (int i = 0; i < population.size(); i++) {
		if (i % GENERATION_SIZE == 0) {
			cout << "\n\nNODE\n";
		}
		cout << population[i] << " ";
	}

	double tt = ComputeTimeEnd();
	cout << "\nGPU time: " << tt << " ms\n";
}

void cpu()
{
	ComputeTimeStart();
	for (int node = 0; node < N; node++) {
		for (int individualNo = 0; individualNo < GENERATION_SIZE; individualNo++) {
			individuals[node][individualNo] = round(rnd() * start_xd_min[node]);
		}
	}

	for (int node = 0; node < N; node++) {
		cout << "\nNODE\n";
		for (int individualNo = 0; individualNo < GENERATION_SIZE; individualNo++) {
			cout << individuals[node][individualNo] << " ";
		}
		cout << "\n";
	}

	double tt = ComputeTimeEnd();
	cout << "CPU time: " << tt << " ms\n";
}

int main()
{
	cpu();
	gpu();

	return 0;
}