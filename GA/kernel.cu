#include "hip/hip_runtime.h"
// thrust library
#include <thrust/device_vector.h>

using namespace std;

const unsigned int N = 3;
const unsigned int GENERATION_SIZE = 100;
const unsigned int start_xd_min[N] = { 2112,1180,417 };
unsigned int individuals[N][GENERATION_SIZE] = {};

#include "cpu_rnd.cuh"
#include "cuda_rnd.cuh"

#include "compute_time.cuh"

void gpu()
{
	// device storage for doubles
	thrust::device_vector<double> population(N * GENERATION_SIZE);

	// Fill the vector with random distribution (0,1)
	thrust::counting_iterator<unsigned int> index_sequence_begin(0);
	thrust::transform(index_sequence_begin,
		index_sequence_begin + GENERATION_SIZE * N,
		population.begin(),
		cuda_rnd());

	ComputeTimeStart();
	for (int node = 0; node < N; node++) {
		for (int individualNo = 0; individualNo < GENERATION_SIZE; individualNo++) {
			int index = individualNo + node * GENERATION_SIZE;
			population[index] = round(population[index] * start_xd_min[node]);
		}
	}

	// print contents of population
	for (int i = 0; i < population.size(); i++) {
		if (i % GENERATION_SIZE == 0) {
			cout << "\n\nNODE\n";
		}
		cout << population[i] << " ";
	}

	double tt = ComputeTimeEnd();
	cout << "\nGPU time: " << tt << " ms\n";
}

void cpu()
{
	ComputeTimeStart();
	for (int node = 0; node < N; node++) {
		for (int individualNo = 0; individualNo < GENERATION_SIZE; individualNo++) {
			individuals[node][individualNo] = round(rnd() * start_xd_min[node]);
		}
	}

	for (int node = 0; node < N; node++) {
		cout << "\nNODE\n";
		for (int individualNo = 0; individualNo < GENERATION_SIZE; individualNo++) {
			cout << individuals[node][individualNo] << " ";
		}
		cout << "\n";
	}

	double tt = ComputeTimeEnd();
	cout << "CPU time: " << tt << " ms\n";
}

int main()
{
	cpu();
	gpu();

	return 0;
}