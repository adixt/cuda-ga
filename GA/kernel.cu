#include "hip/hip_runtime.h"
// thrust library
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
using namespace std;

const unsigned int N = 3;
const unsigned int GENERATION_SIZE = 100;
const unsigned int start_xd_min[N] = { 2112,1180,417 };
unsigned int individuals[N][GENERATION_SIZE] = {};

#include "cpu_rnd.cuh"
#include "cuda_rnd.cuh"

#include "compute_time.cuh"

const int n = 3;
const int ns = 2;
const int allNodes = n + ns;
int LT[allNodes][n] = {};

const int x_inf = 100000;
const int simTime = 60;

void gpu()
{
	// device storage for doubles
	thrust::device_vector<double> population(N * GENERATION_SIZE);

	// Fill the vector with random distribution (0,1)
	thrust::counting_iterator<unsigned int> index_sequence_begin(0);
	thrust::transform(index_sequence_begin,
		index_sequence_begin + GENERATION_SIZE * N,
		population.begin(),
		cuda_rnd());


	thrust::device_vector<int> LTcuda(allNodes * n);

	thrust::copy(&(LT[0][0]), &(LT[allNodes - 1][n - 1]), LTcuda.begin());
	thrust::device_vector<int>::iterator iter =
		thrust::max_element(LTcuda.begin(), LTcuda.end());
	int Lcuda = *iter;

	ComputeTimeStart();
	for (int node = 0; node < N; node++) {
		for (int individualNo = 0; individualNo < GENERATION_SIZE; individualNo++) {
			int index = individualNo + node * GENERATION_SIZE;
			population[index] = round(population[index] * start_xd_min[node]);
		}
	}

	// print contents of population
	for (int i = 0; i < population.size(); i++) {
		if (i % GENERATION_SIZE == 0) {
			cout << "\n\nNODE\n";
		}
		cout << population[i] << " ";
	}

	double tt = ComputeTimeEnd();
	cout << "\nGPU time: " << tt << " ms\n";
}

#include "vector_types.cuh"
void cpu()
{
	ComputeTimeStart();
	for (int node = 0; node < N; node++) {
		for (int individualNo = 0; individualNo < GENERATION_SIZE; individualNo++) {
			individuals[node][individualNo] = round(rnd() * start_xd_min[node]);
		}
	}

	for (int node = 0; node < N; node++) {
		cout << "\nNODE\n";
		for (int individualNo = 0; individualNo < GENERATION_SIZE; individualNo++) {
			cout << individuals[node][individualNo] << " ";
		}
		cout << "\n";
	}

	LT[3][0] = 2;
	LT[4][1] = 4;
	LT[1][2] = 3;
	LT[0][2] = 3;
	LT[0][1] = 1;

	int* start = &LT[0][0];
	// max lead time
	int L = *max_element(start, start + allNodes * n);

	float LA_nom[allNodes][n] = {};
	three_dimension_vector_float LA(simTime, vector<vector<float>>(allNodes, vector<float>(n)));

	LA[0][3][0] = LA_nom[3][0] = 1;
	LA[0][4][1] = LA_nom[4][1] = 0.8f;
	LA[0][1][2] = LA_nom[1][2] = 0.6f;
	LA[0][0][2] = LA_nom[0][2] = 0.4f;
	LA[0][0][1] = LA_nom[0][1] = 0.2f;

	cout << endl;
	for (int i = 0; i < allNodes; i++) {
		for (int j = 0; j < n; j++)
			cout << LA[0][i][j] << " ";
		cout << endl;
	}cout << endl;

	// Verify if allocation correct - elements in each column should sum up to 1 or 0	
	for (int j = 0; j < n; j++) {
		float temp = 0;
		for (int i = 0; i < allNodes; i++) {
			temp = temp + LA[0][i][j];
		}

		if (temp != 0 && temp != 1) {
			throw ("Improper allocation in column: %d", j);
		}
	}

	double tt = ComputeTimeEnd();
	cout << "CPU time: " << tt << " ms\n";
}

int main()
{
	cpu();
	gpu();

	return 0;
}